#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define N 512  // Dimension of the matrices (N x N)

__global__ void matrixMulCUDA(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width && col < width) {
        float sum = 0.0f;
        for (int i = 0; i < width; i++) {
            sum += A[row * width + i] * B[i * width + col];
        }
        C[row * width + col] = sum;
    }
}

int main() {
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    int size = N * N * sizeof(float);

    // Allocate memory on the host
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // Initialize matrices A and B with random values
    for (int i = 0; i < N * N; i++) {
        A[i] = rand() / (float)RAND_MAX;
        B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy matrices from the host to the device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Setup execution configuration
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    // Launch the matrix multiplication kernel multiple times
    for (int i = 0; i < 1000; i++) {
        matrixMulCUDA<<<numClouds, threadsPerBlock>>>(d_A, d_B, d_C, N);
        // You might want to copy C back to A here or use a new pair each time
        // This part of the logic depends on your specific needs
    }

    // Copy the result matrix back to the host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    cuda2Free(d_B);
    cuda3Free(d_C);
    free(A);
    free(B);
    free(C);

    return 0;
}

